#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <iostream>
#include <fstream>
#include <chrono>

#define TYP int

int main()
{
	const int bufSize{ 50'000'000 };

	std::ifstream fileInputStream;
	fileInputStream.open("1.bin", std::ios::in | std::ios::binary);

	if (!fileInputStream.is_open())
		return -1;

	TYP* pBuffer;

	try {
		pBuffer = new TYP[bufSize];
	}
	catch (...) {
		return -2;
	}
	fileInputStream.seekg(9'000'000, std::ios::beg); //avoiding zero-area of file
	fileInputStream.read((char*)pBuffer, bufSize * sizeof(TYP));
	fileInputStream.close();

	if (fileInputStream.bad())
		return -4;

	thrust::host_vector<TYP> h_vec(pBuffer, pBuffer + bufSize);

	delete[] pBuffer;

	auto t1 = std::chrono::high_resolution_clock::now();
	thrust::device_vector<int> d_vec = h_vec;

	auto t2 = std::chrono::high_resolution_clock::now();
	thrust::sort(d_vec.begin(), d_vec.end()); //The main Operation

	auto t3 = std::chrono::high_resolution_clock::now();
	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	auto t4 = std::chrono::high_resolution_clock::now();
	
	const std::chrono::duration<double, std::milli> passed1 = t2 - t1;
	const std::chrono::duration<double, std::milli> passed2 = t3 - t2;
	const std::chrono::duration<double, std::milli> passed3 = t4 - t3;
	
	size_t tcopy{ (size_t)passed1.count() + (size_t)passed3.count() };
	size_t tsort{ (size_t)passed2.count() };
		
		
	std::cout << "Size of Vector = " << h_vec.size() << ", Time elapsed = ";
	std::cout  << tsort << " ms + " << tcopy << " ms" << std::endl;
}
